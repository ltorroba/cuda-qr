#include "hip/hip_runtime.h"
#include "reference_kernels.cuh"
#include <hipblas.h>
#include <hipsolver.h>
#include "cuda_utils.cuh"

void* setup_QRX(const float* A, const float* B, int m, int n, int r) {
    // Allocate memory for Q and R
    size_t total_size = (m * n + n * n) * sizeof(float);
    void* data;
    CHECK_CUDA(hipMalloc(&data, total_size));
    
    // First compute AB
    float* AB;
    CHECK_CUDA(hipMalloc(&AB, m * n * sizeof(float)));
    
    // Use cuBLAS to compute AB
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    const float alpha = 1.0f;
    const float beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n, r,              // dimensions
                &alpha,
                A, m,                 // matrix A (m x r)
                B, r,                 // matrix B (r x n)
                &beta,
                AB, m));              // output AB (m x n)
    
    // Setup cuSolver for QR
    hipsolverHandle_t solver_handle;
    CHECK_CUSOLVER(hipsolverDnCreate(&solver_handle));
    
    // Query working space for QR
    int lwork;
    float* tau;
    CHECK_CUDA(hipMalloc(&tau, n * sizeof(float)));
    CHECK_CUSOLVER(hipsolverDnSgeqrf_bufferSize(solver_handle, m, n, AB, m, &lwork));
    
    // Allocate working space
    float* workspace;
    CHECK_CUDA(hipMalloc(&workspace, lwork * sizeof(float)));
    int* devInfo;
    CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    
    // Compute QR factorization in-place in AB
    CHECK_CUSOLVER(hipsolverDnSgeqrf(solver_handle, m, n,
                     AB, m,
                     tau, workspace, lwork,
                     devInfo));
    
    // Copy R from upper triangular part of AB
    float* R = static_cast<float*>(data) + m * n;  // R starts after Q
    CHECK_CUDA(hipMemset(R, 0, n * n * sizeof(float)));
    for(int j = 0; j < n; j++) {
        CHECK_CUDA(hipMemcpy(R + j * n, AB + j * m, (j + 1) * sizeof(float), hipMemcpyDeviceToDevice));
    }
    
    // Compute explicit Q matrix
    CHECK_CUSOLVER(hipsolverDnSorgqr(solver_handle, m, n, n,
                     AB, m,
                     tau, workspace, lwork,
                     devInfo));
    
    // Copy Q to output data
    float* Q = static_cast<float*>(data);  // Q comes first in data
    CHECK_CUDA(hipMemcpy(Q, AB, m * n * sizeof(float), hipMemcpyDeviceToDevice));
    
    // Cleanup
    CHECK_CUDA(hipFree(AB));
    CHECK_CUDA(hipFree(tau));
    CHECK_CUDA(hipFree(workspace));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUSOLVER(hipsolverDnDestroy(solver_handle));
    CHECK_CUBLAS(hipblasDestroy(handle));
    
    return data;
}

__global__ void compute_QRx(const void* kernel_data, const float* x, float* output, int m, int n, int r) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m) return;

    const float* Q = static_cast<const float*>(kernel_data);
    const float* R = Q + m * n;

    float sum = 0.0f;
    for (int j = 0; j < r; j++) {
        float QR_row_j = 0.0f;
        for (int k = 0; k < n; k++) {
            QR_row_j += Q[row + k * m] * R[k + j * n];
            printf("Q[%d] = %f\n", row + k * m, Q[row + k * m]);
            printf("R[%d] = %f\n", k + j * n, R[k + j * n]);
        }
        printf("x[%d] = %f\n", j, x[j]);
        sum += QR_row_j * x[j];
    }
    output[row] = sum;
}

__global__ void compute_QRX(const void* kernel_data, const float* X, float* output,
                           int m, int n, int r, int k) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row >= m || col >= k) return;

    const float* Q = static_cast<const float*>(kernel_data);
    const float* R = Q + m * n;
    
    float sum = 0.0f;
    for (int j = 0; j < r; j++) {
        float QR_row_j = 0.0f;
        for (int i = 0; i < n; i++) {
            QR_row_j += Q[row + i * m] * R[i + j * n];
        }
        sum += QR_row_j * X[j + col * r];
    }
    output[row + col * m] = sum;
}

// Launch function implementations
void launch_QRx(const void* kernel_data, const float* x, float* output, 
                int m, int n, int r) {
    // Choose block size and compute grid size
    const int BLOCK_SIZE = 256;
    dim3 block(BLOCK_SIZE);
    dim3 grid((m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Launch kernel
    compute_QRx<<<grid, block>>>(kernel_data, x, output, m, n, r);
    CHECK_KERNEL();
    CHECK_CUDA(hipDeviceSynchronize());
}

void launch_QRX(const void* kernel_data, const float* X, float* output,
                int m, int n, int r, int k) {
    // Choose block dimensions
    const int BLOCK_DIM_X = 16;
    const int BLOCK_DIM_Y = 16;
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid((m + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
              (k + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);
    
    // Launch kernel
    compute_QRX<<<grid, block>>>(kernel_data, X, output, m, n, r, k);
    CHECK_KERNEL();
    CHECK_CUDA(hipDeviceSynchronize());
}

void* setup_ABX(const float* A, const float* B, int m, int n, int r) {
    size_t total_size = (m * r + r * n) * sizeof(float);
    float* data;
    CHECK_CUDA(hipMalloc(&data, total_size));

    // Copy A and B to data
    CHECK_CUDA(hipMemcpy(data, A, m * r * sizeof(float), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(data + m * r, B, r * n * sizeof(float), hipMemcpyDeviceToDevice));

    return data;
}

void cublas_ABx(const void* kernel_data, const float* x, float* output,
                int m, int n, int r) {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float* temp;
    CHECK_CUDA(hipMalloc(&temp, n * sizeof(float)));

    const float alpha = 1.0f;
    const float beta = 0.0f;

    const float* A = static_cast<const float*>(kernel_data);
    const float* B = A + m * r;

    // First compute Bx = temp
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N,
                n, r,                  // dimensions
                &alpha,
                B, n,                  // matrix B (n x r)
                x, 1,                  // vector x (r x 1)
                &beta,
                temp, 1));            // output temp (n x 1)

    // Then compute A(Bx) = output
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N,
                m, n,                  // dimensions
                &alpha,
                A, m,                  // matrix A (m x n)
                temp, 1,               // vector temp (n x 1)
                &beta,
                output, 1));          // output (m x 1)

    // Cleanup
    CHECK_CUDA(hipFree(temp));
    CHECK_CUBLAS(hipblasDestroy(handle));
}

void cublas_ABX(const void* kernel_data, const float* X, float* output,
                int m, int n, int r, int k) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    float* temp;
    CHECK_CUDA(hipMalloc(&temp, n * sizeof(float)));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    const float* A = static_cast<const float*>(kernel_data);
    const float* B = A + m * r;

    // First compute BX = temp
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, k, r,              // dimensions
                &alpha,
                B, n,                  // matrix B (n x r)
                X, r,                  // matrix X (r x k)
                &beta,
                temp, n));             // output temp (n x k)

    // Then compute A(BX) = output
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, k, n,              // dimensions
                &alpha,
                A, m,                  // matrix A (m x n)
                temp, n,               // matrix temp (n x k)
                &beta,
                output, m));           // output (m x k)
} 