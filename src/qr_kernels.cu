#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <map>
#include <random>
#include <tuple>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

__device__ inline void cp_async4(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 16;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.cg.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}
__device__ inline void cp_async1(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 4;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.ca.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}

__device__ __forceinline__ void async_memcpy_waitall() {
    asm volatile("cp.async.wait_all;\n" ::);
}

////////////////////////////////////////////////////////////////////////////////



namespace qr_base {

#define tilesize 32
#define numthreads 4


//-------------------- multiply by Q kernels-----------------------------------

__global__ void base_applyQt_singletile_evelyne( //aplies Qt (given by householder reflectors on diagonal tile k) to the remainder of the row
    int size_in,
    int diag_iter,
    float const *tau,
    float *out) {
    int g = blockIdx.x;
    int i = threadIdx.x;
    int j = threadIdx.y;
    __shared__ float outs[tilesize][tilesize];
    __shared__ float Qs[tilesize][tilesize];
    __shared__ float cache[tilesize][numthreads];
    int diagstartidx=diag_iter*tilesize;
    int tileoffset=(1+g)*tilesize;
    
    
    for (int l=j;l<tilesize;l+=numthreads){
        outs[i][l]=out[(i+diagstartidx)*size_in+l+diagstartidx+tileoffset];
        Qs[i][l]=out[(i+diagstartidx)*size_in+l+diagstartidx];
    }
    

    __syncthreads();

    for (int k=0;k<tilesize-1;k++){
        float tmp_sum = 0.0f;
        for (int l=k+j+1;l<tilesize;l+=numthreads){
            tmp_sum+= Qs[l][k]*outs[l][i];
        }
        cache[i][j]=tmp_sum;
        __syncthreads();
        tmp_sum=outs[k][i];
        for (int l=0;l<numthreads;l++){
            tmp_sum+=cache[i][l];
        }
        tmp_sum*=tau[(diag_iter)*size_in+k];
        for (int l=k+j+1;l<tilesize;l+=numthreads){
            outs[l][i]-=tmp_sum*Qs[l][k];
        }
        if (j==0){
            outs[k][i]-=tmp_sum;
        }
        __syncthreads();
    }

    for (int l=j;l<tilesize;l+=numthreads){
        out[(i+diagstartidx)*size_in+l+diagstartidx+tileoffset]=outs[i][l];
    }
}

//// LUCAS' ADDITIONS---NEED TO MERGE PROPERLY
__host__ __device__ __forceinline__ int32_t ceil_div(int32_t a, int32_t b) { return (a + b - 1) / b; }
constexpr int32_t __host__ __device__ ceil_div_static(int32_t a, int32_t b) { return (a + b - 1) / b; }

template <typename T>
__host__ __device__ __forceinline__ void swap_pointers(T** a, T** b) {
    auto temp_a = *a;
    *a = *b;
    *b = temp_a;
}

// This applies Q'X to the remainder of the row
__global__ void base_applyQt_singletile(int size_in, int diag_iter, float const *tau, float *out) {
    // TODO: Make template
    auto const tile_size = tilesize;

    auto num_threads = gridDim.x * blockDim.x;
    auto thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto columns_to_skip = (diag_iter + 1) * tilesize;
    auto columns_per_thread = ceil_div(size_in - columns_to_skip, num_threads);

    // TODO: Use all threads in block to load the columns we will be processing using a better
    //       access pattern

    // Load householder reflectors and taus
    float householder_reflectors[tile_size][tile_size];
    float taus[tile_size];
    for (auto reflector_idx = 0; reflector_idx < tile_size; reflector_idx++) {
        taus[reflector_idx] = tau[diag_iter * size_in + reflector_idx];
        for (auto element_idx = reflector_idx + 1; element_idx < tile_size; element_idx++) {
            auto householder_reflector_i = diag_iter * tile_size + element_idx;
            auto householder_reflector_j = diag_iter * tile_size + reflector_idx;
            householder_reflectors[element_idx][reflector_idx] = out[householder_reflector_i * size_in + householder_reflector_j];
        }
    }

    float current_column[tile_size];
    for (auto current_column_idx = 0; current_column_idx < columns_per_thread; current_column_idx++) {
        auto current_column_j = columns_to_skip + thread_idx * columns_per_thread + current_column_idx;

        if (current_column_j >= size_in)
            break;

        // Load current column we are processing
        for (auto local_i = 0; local_i < tilesize; local_i++) {
            auto current_column_i = diag_iter * tile_size + local_i;
            current_column[local_i] = out[current_column_i * size_in + current_column_j];
        }

        // Process current column by applying householder reflectors in reverse order
        for (auto householder_reflector = 0; householder_reflector < tile_size; householder_reflector++) {
            // First we compute tau * (h' x)
            auto effective_scaling = 0.0f;
            for (auto element_idx = 0; element_idx < tile_size; element_idx++) {
                if (element_idx == householder_reflector) {
                    // Implicit leading 1 in householder reflector
                    effective_scaling += current_column[element_idx];
                } else if (element_idx > householder_reflector) {
                    effective_scaling += householder_reflectors[element_idx][householder_reflector] * current_column[element_idx];
                }
            }
            effective_scaling *= taus[householder_reflector];

            // We now compute h (tau * (h' x)) to wrap things up
            for (auto element_idx = 0; element_idx < tile_size; element_idx++) {
                if (element_idx == householder_reflector) {
                    current_column[element_idx] -= effective_scaling;
                } else if (element_idx > householder_reflector) {
                    current_column[element_idx] -= effective_scaling * householder_reflectors[element_idx][householder_reflector];
                }
            }
        }

        // Write out processed column
        for (auto local_i = 0; local_i < tile_size; local_i++) {
            auto current_column_i = diag_iter * tile_size + local_i;
            out[current_column_i * size_in + current_column_j] = current_column[local_i];
        }
    }
}

void launch_base_applyQt_singletile(int size_in, int diag_iter, float const *tau, float *out) {
    // base_applyQt_singletile_evelyne<<<1, dim3(tilesize, numthreads)>>>(size_in, diag_iter, tau, out); 
    base_applyQt_singletile<<<tilesize, tilesize>>>(size_in, diag_iter, tau, out); 
}

////// END LUCAS' ADDITIONS

__global__ void base_applyQt_doubletile( //aplies Qt (given by householder reflectors on the tile at row_idx below diag_idx) to the remainder of the row, and to the row of diag_idx
    int size_in,
    int diag_iter,
    int row_iter,
    float const *tau,
    float *out) {
    int g = blockIdx.x;
    int i = threadIdx.x;
    int j = threadIdx.y;
    __shared__ float outs[2*tilesize][tilesize];
    __shared__ float Qs[tilesize][tilesize];
    __shared__ float cache[tilesize][numthreads];
    int diagstartidx=diag_iter*tilesize;
    int tileoffset=(1+g)*tilesize;
    int iteroffset=row_iter*tilesize;
    
    
    for (int l=j;l<tilesize;l+=numthreads){
        outs[i][l]=out[(i+diagstartidx)*size_in+l+diagstartidx+tileoffset];
        outs[i+tilesize][l]=out[(i+diagstartidx+iteroffset)*size_in+l+diagstartidx+tileoffset];
        Qs[i][l]=out[(i+diagstartidx+iteroffset)*size_in+l+diagstartidx];
    }


    __syncthreads();

    for (int k=0;k<tilesize;k++){
        float tmp_sum = 0.0f;
        for (int l=j;l<tilesize;l+=numthreads){
            tmp_sum+= Qs[l][k]*outs[l+tilesize][i];
        }
        cache[i][j]=tmp_sum;
        __syncthreads();
        tmp_sum=outs[k][i];
        for (int l=0;l<numthreads;l++){
            tmp_sum+=cache[i][l];
        }
        tmp_sum*=tau[(diag_iter)*size_in+row_iter*tilesize+k];
        if (j==0){
            outs[k][i]-=tmp_sum;
        }
        for (int l=j;l<tilesize;l+=numthreads){
            outs[l+tilesize][i]-=tmp_sum*Qs[l][k];
        }
        
        __syncthreads();
    }

    for (int l=j;l<tilesize;l+=numthreads){
        out[(i+diagstartidx)*size_in+l+diagstartidx+tileoffset]=outs[i][l];
        out[(i+diagstartidx+iteroffset)*size_in+l+diagstartidx+tileoffset]=outs[i+tilesize][l];
    }

}
//-------------------- calculate QR kernels-----------------------------------


__global__ void base_calcQR_singletile( //calculates in-place QR of diagonal tile
    int size_in,
    int diag_iter,
    float *tau,
    float *out) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    __shared__ float outs[tilesize][tilesize];
    __shared__ float cache[tilesize];
    __shared__ float tauvals[2];
    int diagstartidx=diag_iter*tilesize;

    outs[i][j]=out[(i+diagstartidx)*size_in+j+diagstartidx];
    __syncthreads();

    for(int iter=0;iter<tilesize-1;iter++){
        if (i>iter && j==0){
            cache[i]=outs[i][iter]*outs[i][iter];
        } 
        
        __syncthreads();
        if (i==0 && j==0){
            float tmp_sum=0.0f;
            for (int l=iter+1;l<tilesize;l++){
                tmp_sum+=cache[l];
             }
            float tmp_sum2=sqrt(tmp_sum+pow(outs[iter][iter],2));
            float newvalue=outs[iter][iter];
            if (newvalue>0){
                newvalue+=tmp_sum2;
            }else{
                newvalue-=tmp_sum2;
            }
            tmp_sum2=sqrt(tmp_sum+pow(newvalue,2));
            tauvals[0]=2 * pow(newvalue/tmp_sum2,2);
            tauvals[1]= newvalue;
            tau[(diag_iter)*size_in+iter]=tauvals[0];

        }
        float tmp_sum=0.0f;
        if (j>=iter && i>=iter){
            for (int k=iter+1;k<tilesize;k++){
                tmp_sum+=outs[k][iter]*outs[k][j];
            }
        }
        float tileiterj=outs[iter][j];
        float tileiiter = outs[i][iter];
        __syncthreads();
        if (j>=iter && i>=iter){
            tmp_sum = (tmp_sum / tauvals[1]+tileiterj)*tauvals[0];
            
            tileiiter/=tauvals[1];

            if (j==iter && i>iter){
                outs[i][j]=tileiiter;
            }else if(i>iter){
                outs[i][j]-=tileiiter*tmp_sum;
            }else{
                outs[i][j]-=tmp_sum;
            }
        }
        __syncthreads();
        out[(i+diagstartidx)*size_in+j+diagstartidx]=outs[i][j];
    }

}

__global__ void base_calcQR_doubletile( //calculates in-place QR of diagonal tile combined with row_idx tile below
    int size_in,
    int diag_iter,
    int row_iter,
    float *tau,
    float *out) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    __shared__ float outs[2*tilesize][tilesize];
    __shared__ float cache[2*tilesize];
    __shared__ float tauvals[2];
    int diagstartidx=diag_iter*tilesize;
    int iteroffset=row_iter*tilesize;

    outs[i][j]=out[(i+diagstartidx)*size_in+j+diagstartidx];
    outs[i+tilesize][j]=out[(i+diagstartidx+iteroffset)*size_in+j+diagstartidx];
    
    for(int iter=0;iter<tilesize;iter++){
        if (j==iter){
            cache[i]=outs[i+tilesize][iter]*outs[i+tilesize][iter];
        }
        __syncthreads();
        if (i==0 && j==0){
            float tmp_sum=0.0f;
            for (int l=0;l<tilesize;l++){
                tmp_sum+=cache[l];
            }
            float tmp_sum2=sqrt(tmp_sum+outs[iter][iter]*outs[iter][iter]);
            float newvalue=outs[iter][iter];
            if (newvalue>0){
                newvalue+=tmp_sum2;
            }else{
                newvalue-=tmp_sum2;
            }
            tmp_sum2=sqrt(tmp_sum+newvalue*newvalue);
            tauvals[0]=2 * (newvalue/tmp_sum2)*(newvalue/tmp_sum2);
            tauvals[1]= newvalue;
            tau[(diag_iter)*size_in+row_iter*tilesize+iter]=tauvals[0];
        }
        float tileiterj=outs[iter][j];
        float tileiiter = outs[i+tilesize][iter];
        float tmp_sum=0.0f;
        if (j>=iter){
            for (int k=tilesize;k<tilesize*2;k++){
                tmp_sum+=outs[k][iter]*outs[k][j];
            }
        }

        __syncthreads();
        if (j>=iter){ //j 0,1
            tmp_sum += ( tauvals[1]*tileiterj);
            if (i==iter){ //i 0
                outs[i][j]-=tmp_sum*tauvals[0]/tauvals[1];
            }
            if(j>iter){// i 1
                outs[i+tilesize][j]-=tileiiter*tmp_sum * tauvals[0]/tauvals[1]/tauvals[1];
            }
            
    

        }
        __syncthreads();
        if (j==0){ //j 0 i 0 1
            outs[i+tilesize][iter]=tileiiter / tauvals[1];
        }
        __syncthreads();

    }
    __syncthreads();
    out[(i+diagstartidx)*size_in+j+diagstartidx]=outs[i][j];
    out[(i+diagstartidx+iteroffset)*size_in+j+diagstartidx]=outs[i+tilesize][j];


}



        

void launch_tiled_qr(
    int32_t size_i,
    float *a, float *tau) {
        /*
    if ( numthreadsperblockb %4 !=0 || nummemperblock %4!=0 || ilpnuma%4!=0 || ilpnumb%4!=0){
            throw std::invalid_argument( "Not implemented for this argument size" );
    }
    int result=0;
    cudaOccupancyMaxActiveBlocksPerMultiprocessor (&result,matmul_improved_macro2, numthreadsperblocka*numthreadsperblockb, 0);
    printf("%d\n",result);
    //uint32_t shmem_size_bytes = (((numthreadsperblocka)*(nummemperblock+1)*ilpnuma+(numthreadsperblockb)*(nummemperblock)*ilpnumb));
    int32_t noblocksa=(size_i+numthreadsperblocka*ilpnuma-1)/(numthreadsperblocka*ilpnuma);
    int32_t noblocksb=((size_j+numthreadsperblockb*ilpnumb-1))/(numthreadsperblockb*ilpnumb);
    dim3 num_blocks = dim3(noblocksa*noblocksb,1,1  );
    dim3 block_size = dim3(numthreadsperblocka*numthreadsperblockb,1,1);
    //CUDA_CHECK(cudaFuncSetAttribute( matmul_improved_macro2, cudaFuncAttributeMaxDynamicSharedMemorySize, shmem_size_bytes));
    matmul_improved_macro2<<<num_blocks, block_size>>>(size_i,size_j,size_k,noblocksa,noblocksb,a,b,c);
        */
    }

    void test_qrkernel_single(
        int32_t size_i,
        float *a, float *tau) {
        base_calcQR_singletile<<<1,dim3(tilesize,tilesize)>>>(size_i,0,tau,a);
    
    
        }

    void test_mulqtkernel_single(
        int32_t size_i,
        float *a, float *tau) {
            base_calcQR_singletile<<<1,dim3(tilesize,tilesize)>>>(size_i,0,tau,a); 
            launch_base_applyQt_singletile(size_i, 0, tau, a);


    
        }
        void test_qrkernel_double(
            int32_t size_i,
            float *a, float *tau) {
            base_calcQR_singletile<<<1,dim3(tilesize,tilesize)>>>(size_i,0,tau,a); 
            base_calcQR_doubletile<<<1,dim3(tilesize,tilesize)>>>(size_i,0,1,tau,a); 
        
        
            }
    
        void test_mulqtkernel_double(
            int32_t size_i,
            float *a, float *tau) {
                base_calcQR_singletile<<<1,dim3(tilesize,tilesize)>>>(size_i,0,tau,a); 
                launch_base_applyQt_singletile(size_i, 0, tau, a);
                base_calcQR_doubletile<<<1,dim3(tilesize,tilesize)>>>(size_i,0,1,tau,a);
                base_applyQt_doubletile<<<1,dim3(tilesize,numthreads)>>>(size_i,0,1,tau,a); 
                base_calcQR_singletile<<<1,dim3(tilesize,tilesize)>>>(size_i,1,tau,a); 
        
            }
        


};

////////////////////////////////////////////////////////////////////////////////

void print_matrix(int32_t n_row, int32_t n_col, std::vector<float> const &matrix) {
    for (int32_t i = 0; i < n_row; i++) {
        printf("    ");
        for (int32_t j = 0; j < n_col; j++) {
            printf("%10.5f ", matrix.at(i * n_col + j));
        }
        printf("\n");
    }
}

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename Reset, typename F>
double
benchmark_ms(double target_time_ms, int32_t num_iters_inner, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
    }
    return best_time_ms;
}

struct BenchmarkConfig {
    int32_t size_i;
    int32_t size_j;
};

struct TestData {
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> a;
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> ref;
};

TestData read_test_data(
    std::string const &test_data_dir,
    std::vector<BenchmarkConfig> const &configs) {
    auto data = TestData{};
    for (auto const &config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto path_prefix = test_data_dir + "/test_";

        if (data.a.find({size_i, size_j}) == data.a.end()) {
            data.a[{size_i, size_j}] = read_data(
                path_prefix + "a_" + std::to_string(size_i) + "_" +
                    std::to_string(size_j) + ".bin",
                size_i * size_j);
        }

        if (data.ref.find({size_i, size_j}) == data.ref.end()) {
            data.ref[{size_i, size_j}] = read_data(
                path_prefix + "ref_" + std::to_string(size_i) + "_" +
                    std::to_string(size_j) + ".bin",
                size_i * size_j);
        }

    }
    return data;
}

struct BenchmarkResults {
    char const *name;
    std::map<std::tuple<int32_t, int32_t>, double> elapsed_ms;
};

enum class Phase {
    TEST,
    WARMUP,
    BENCHMARK,
};

template <typename Impl>
void run_config(
    Phase phase,
    TestData const &data,
    BenchmarkConfig const &config,
    BenchmarkResults &results) {
    auto size_i = config.size_i;
    auto size_j = config.size_j;

    auto const &a = data.a.at({size_i, size_j});
    auto const &ref = data.ref.at({size_i, size_j});

    float *a_gpu;
    float *tau_gpu;
    CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_j * sizeof(float)));
    CUDA_CHECK(hipMalloc(&tau_gpu, size_i * size_j * sizeof(float))); // TODO: determine size more accurately

    CUDA_CHECK(hipMemcpy(
        a_gpu,
        a.data(),
        size_i * size_j * sizeof(float),
        hipMemcpyHostToDevice));


    size_t workspace_size = Impl::get_workspace_size(size_i, size_j);
    float *workspace_gpu = nullptr;
    if (workspace_size > 0) {
        CUDA_CHECK(hipMalloc(&workspace_gpu, workspace_size));
        CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
    }

    if (phase == Phase::BENCHMARK) {
        printf("  %6d  %6d  ", size_i, size_j);
    } else {
        printf("  warmup %6d  %6d", size_i, size_j);
    }

    if (phase == Phase::TEST){
        if(size_i==tilesize && size_j==tilesize){
            printf(" %-12s ", "(testqr)");
            Impl::testqr(size_j, a_gpu, tau_gpu);
        }else if (size_i==tilesize){
            printf(" %-12s ", "(testmulq)");
            Impl::testmulq(size_j, a_gpu, tau_gpu);
        }else if (size_j==tilesize){
            printf(" %-12s ", "(testqr2)");
            Impl::testqr2(size_j, a_gpu, tau_gpu);
        }else{
            printf(" %-12s ", "(testmulq2)");
            Impl::testmulq2(size_j, a_gpu, tau_gpu);
        }

    }else{
        Impl::run(size_j,   a_gpu, tau_gpu);
    }

    std::vector<float> c_out_host(size_i * size_j);
    CUDA_CHECK(hipMemcpy(
        c_out_host.data(),
        a_gpu,
        size_i * size_j * sizeof(float),
        hipMemcpyDeviceToHost));

    double mse = 0.0;
    double ref_mean_square = 0.0;
    for (int32_t i = 0; i < size_i; ++i) {
        for (int32_t j = i; j < size_j; ++j) {
            float diff = abs(c_out_host[i * size_j + j]) - abs(ref[i * size_j + j]);
            mse += diff * diff;
            ref_mean_square += abs(ref[i * size_j + j]) * abs(ref[i * size_j + j]);
        }
    }
    mse /= (size_i * size_j/2);
    ref_mean_square /= (size_i * size_j/2);
    float rmse = std::sqrt(mse);
    float rel_rmse = rmse / std::sqrt(ref_mean_square);

    if (phase == Phase::BENCHMARK || phase == Phase::TEST ) {
        printf("  -- rmse:  %8.02e", rel_rmse);
    }

    if (rel_rmse > 1e-3) {
        if (phase == Phase::BENCHMARK) {
            printf("  %9s  %7s", "-", "-");
        } else if (phase == Phase::TEST){
            printf("\n");
            printf("  expected output:\n");
            // print_matrix(size_i, size_j, ref);
            printf("\n");
            printf("  obtained output:\n");
            // print_matrix(size_i,  size_j, c_out_host);
        }
    } else {
        double target_time_ms = 200.0;
        double elapsed_ms = benchmark_ms(
            target_time_ms,
            20,
            [&]() {
                if (workspace_size > 0) {
                    CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
                }
            },
            [&]() {
                if (phase == Phase::TEST){
                    if(size_i==tilesize && size_j==tilesize){
                        Impl::testqr(size_j, a_gpu, tau_gpu);
                    }else if (size_i==tilesize){
                        Impl::testmulq(size_j, a_gpu, tau_gpu);
                    }else if (size_j==tilesize){
                        Impl::testqr2(size_j, a_gpu, tau_gpu);
                    }else{
                        Impl::testmulq2(size_j, a_gpu, tau_gpu);
                    }
                }else{
                    Impl::run(size_j,   a_gpu, tau_gpu);
                }
            });

        results.elapsed_ms[{size_i, size_j}] = elapsed_ms;
        printf("    time:   %8.02e", elapsed_ms);
    }

    printf("\n");

    CUDA_CHECK(hipFree(a_gpu));
    CUDA_CHECK(hipFree(tau_gpu));
    CUDA_CHECK(hipFree(workspace_gpu));
}

template <typename Impl>
BenchmarkResults run_all_configs(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = BenchmarkResults{Impl::name};
    if (phase == Phase::WARMUP) {
        printf("warmup %s:\n\n", Impl::name);
    } else if (phase == Phase::TEST){
        printf("testing %s:\n\n", Impl::name);
    }else {
        printf("%s:\n\n", Impl::name);
        printf(
            "  %-6s  %-6s   %-8s  %-9s  %-7s\n",
            "size_i",
            "size_j",
            "RRMSE",
            "time (ms)",
            "TFLOP/s");
        printf(
            "  %-6s  %-6s  %-8s  %-9s  %-7s\n",
            "------",
            "------",
            "--------",
            "---------",
            "-------");
    }
    for (auto const &config : configs) {
        run_config<Impl>(phase, data, config, results);
    }
    printf("\n");
    return results;
}



struct QRbase {
    constexpr static char const *name = "qr_base";

    static size_t get_workspace_size(int32_t size_i, int32_t size_j) {
        return 0;
    }

    static void
    run(int32_t size_i,
        float *a, float *tau) {
        qr_base::launch_tiled_qr(size_i,a, tau) ;
    }

    static void
    testqr(int32_t size_i,
        float *a, float *tau) {
        qr_base::test_qrkernel_single(size_i,a, tau) ;
    }

    static void
    testqr2(int32_t size_i,
        float *a, float *tau) {
        qr_base::test_qrkernel_double(size_i,a, tau) ;
    }
    static void
    testmulq(int32_t size_i,
        float *a, float *tau) {
        qr_base::test_mulqtkernel_single(size_i,a, tau) ;
    }

    static void
    testmulq2(int32_t size_i,
        float *a, float *tau) {
        qr_base::test_mulqtkernel_double(size_i,a, tau) ;
    }
};



std::vector<BenchmarkResults> run_all_impls(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = std::vector<BenchmarkResults>{};
    results.push_back(run_all_configs<QRbase>(phase, data, configs));
    return results;
}

void write_json_results(
    std::string const &path,
    std::vector<BenchmarkResults> const &results) {
    auto file = std::ofstream(path);
    file << "{\n";
    for (int32_t i = 0; i < results.size(); ++i) {
        auto const &result = results.at(i);
        file << "  \"" << result.name << "\": [\n";
        int32_t j = 0;
        for (auto const &[config, elapsed_ms] : result.elapsed_ms) {
            auto [size_i, size_j] = config;
            double tflop = 2.0 * size_i * size_j * 1e-12;
            double tflop_per_sec = tflop / (elapsed_ms * 1e-3);
            file << "    {\n";
            file << "      \"size_i\": " << size_i << ",\n";
            file << "      \"size_j\": " << size_j << ",\n";
            file << "      \"elapsed_ms\": " << elapsed_ms << ",\n";
            file << "      \"tflop_per_sec\": " << tflop_per_sec << "\n";
            file << "    }";
            if (j + 1 < result.elapsed_ms.size()) {
                file << ",";
            }
            file << "\n";
            ++j;
        }
        file << "  ]";
        if (i + 1 < results.size()) {
            file << ",";
        }
        file << "\n";
    }
    file << "}\n";
}

int main(int argc, char **argv) {
    std::string test_data_dir = ".";
    if (char *c_str_test_data_dir = std::getenv("QR_TEST_DATA_DIR")) {
        test_data_dir = c_str_test_data_dir;
    }

    auto configs_test = std::vector<BenchmarkConfig>{
        {{tilesize,tilesize}, {tilesize*2,tilesize},  {tilesize,tilesize*2}, {tilesize*2,tilesize*2}},
    };

    
    auto data = read_test_data(test_data_dir, configs_test);
    run_all_impls(Phase::TEST, data, configs_test);
    
    /*run_all_impls(Phase::WARMUP, data, configs);
    auto results = run_all_impls(Phase::BENCHMARK, data, configs);

    for (int32_t j = 1; j < results.size(); ++j) {
        for (int32_t i = j; i > 0;) {
            --i;
            auto const &first = results.at(i);
            auto const &second = results.at(j);
            printf("\nspeedups %s -> %s:\n\n", first.name, second.name);
            printf("  %-6s  %-6s  %-6s  %-7s\n", "size_i", "size_j", "size_k", "speedup");
            printf("  %-6s  %-6s  %-6s  %-7s\n", "------", "------", "------", "-------");
            for (auto const &config : configs) {
                auto size_i = config.size_i;
                auto size_j = config.size_j;
                auto size_k = config.size_k;
                printf("  %6d  %6d  %6d", size_i, size_j, size_k);
                auto it_first = first.elapsed_ms.find({size_i, size_j, size_k});
                auto it_second = second.elapsed_ms.find({size_i, size_j, size_k});
                if (it_first != first.elapsed_ms.end() &&
                    it_second != second.elapsed_ms.end()) {
                    printf("  %6.02fx", it_first->second / it_second->second);
                } else {
                    printf("  %7s", "-");
                }
                printf("\n");
            }
        }
            
    }*/

    //write_json_results("out/results.json", results);

    /*auto configs_bench = std::vector<BenchmarkConfig>{
        {{32,32}, {128,128},  {512,512}, {2048,2048}},
    };*/

    return 0;
}
