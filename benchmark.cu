#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <hipsolver.h>
#include "reference_kernels.cuh"

// Add this before main()
template<typename F>
double benchmark_kernel(F func, int num_trials = 1) {
    hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < num_trials; i++) {
        func();
        hipDeviceSynchronize();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / (double)num_trials;
}

// Helper function to generate random low-rank matrix
std::tuple<float*, float*, float*> generateRandomLowRankMatrix(
    hipblasHandle_t handle, hiprandGenerator_t gen, int m, int n, int rank) {
    // Generate A = UV^T where U is m x rank and V is n x rank
    float* d_U;
    float* d_V;
    float* d_A;
    hipMalloc(&d_U, m * rank * sizeof(float));
    hipMalloc(&d_V, n * rank * sizeof(float));
    hipMalloc(&d_A, m * n * sizeof(float));

    // Generate random U and V
    hiprandGenerateNormal(gen, d_U, m * rank, 0.0f, 1.0f);
    hiprandGenerateNormal(gen, d_V, n * rank, 0.0f, 1.0f);

    // Compute A = U * V^T
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                m, n, rank,
                &alpha,
                d_U, m,
                d_V, n,
                &beta,
                d_A, m);

    return {d_A, d_U, d_V};
}

struct CompactQR {
    float* d_A;      // Compact form (m x n)
    float* d_tau;    // Householder scalars (n)
    int m;           // Number of rows
    int n;           // Number of columns
    
    // Destructor to handle cleanup
    ~CompactQR() {
        if (d_A) hipFree(d_A);
        if (d_tau) hipFree(d_tau);
    }
    
    // Prevent copying
    CompactQR(const CompactQR&) = delete;
    CompactQR& operator=(const CompactQR&) = delete;
    
    // Allow moving
    CompactQR(CompactQR&& other) noexcept 
        : d_A(other.d_A), d_tau(other.d_tau), m(other.m), n(other.n) {
        other.d_A = nullptr;
        other.d_tau = nullptr;
    }
    
    // Constructor
    CompactQR(float* a, float* tau, int rows, int cols) 
        : d_A(a), d_tau(tau), m(rows), n(cols) {}
};

std::tuple<float*, float*, CompactQR> computeQR(hipsolverHandle_t solver_handle, 
                                               const float* d_input, int m, int n) {
    // Allocate output matrices
    float* d_Q;
    float* d_R;
    float* d_A;  // For compact form
    hipMalloc(&d_Q, m * n * sizeof(float));
    hipMalloc(&d_R, n * n * sizeof(float));
    hipMalloc(&d_A, m * n * sizeof(float));
    
    // Copy input to compact form
    hipMemcpy(d_A, d_input, m * n * sizeof(float), hipMemcpyDeviceToDevice);
    
    // Allocate workspace for QR decomposition
    int work_size = 0;
    hipsolverDnSgeqrf_bufferSize(solver_handle, m, n, d_A, m, &work_size);
    
    float* d_work;
    float* d_tau;
    int* d_info;
    hipMalloc(&d_work, work_size * sizeof(float));
    hipMalloc(&d_tau, n * sizeof(float));
    hipMalloc(&d_info, sizeof(int));

    // Compute QR factorization (A = QR) in compact form
    hipsolverDnSgeqrf(solver_handle, m, n, d_A, m, d_tau, d_work, work_size, d_info);

    // Copy upper triangular part to R
    hipMemset(d_R, 0, n * n * sizeof(float));
    for(int i = 0; i < n; i++) {
        hipMemcpy(&d_R[i * n], &d_A[i * m], (i + 1) * sizeof(float), 
                  hipMemcpyDeviceToDevice);
    }

    // Compute Q explicitly
    hipMemcpy(d_Q, d_A, m * n * sizeof(float), hipMemcpyDeviceToDevice);
    hipsolverDnSorgqr(solver_handle, m, n, n, d_Q, m, d_tau, d_work, work_size, d_info);

    // Cleanup temporary workspace
    hipFree(d_work);
    hipFree(d_info);
    
    // Create CompactQR struct (transfers ownership of d_A and d_tau)
    CompactQR compact(d_A, d_tau, m, n);
    
    return std::make_tuple(d_Q, d_R, std::move(compact));
}

int main(int argc, char **argv) {
    const int num_trials = 1000;
    const int m = 1024;  // matrix height
    const int n = 512;   // matrix width
    const int r = 32;    // rank
    const int batch_size = 100;  // for matrix X

    // Initialize CUDA resources
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    // Allocate device memory
    float *d_Q, *d_R, *d_x, *d_X, *d_output, *d_output_matrix;
    hipMalloc(&d_Q, m * n * sizeof(float));
    hipMalloc(&d_R, n * n * sizeof(float));
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_X, n * batch_size * sizeof(float));
    hipMalloc(&d_output, m * sizeof(float));
    hipMalloc(&d_output_matrix, m * batch_size * sizeof(float));

    float *d_temp;
    hipMalloc(&d_temp, n * sizeof(float));

    // Benchmark vector multiplication
    double total_time_vector = 0.0;
    for (int trial = 0; trial < num_trials; trial++) {
        // Generate random low-rank matrix
        auto [d_A, d_U, d_V] = generateRandomLowRankMatrix(handle, gen, m, n, r);

        // Compute QR decomposition
        auto [d_Q, d_R, compact] = computeQR(solver_handle, d_A, m, n);
        
        // Generate random vector x
        hiprandGenerateNormal(gen, d_x, n, 0.0f, 1.0f);

        // Time the vector multiplication
        auto kernel_time = benchmark_kernel([&]() {
            launch_QRx(d_Q, d_R, d_x, d_output, m, n, r);
        });
        total_time_vector += kernel_time;

        hipFree(d_A);
        hipFree(d_U);
        hipFree(d_V);
    }

    // Benchmark matrix multiplication
    double total_time_matrix = 0.0;
    for (int trial = 0; trial < num_trials; trial++) {
        // Generate random low-rank matrix
        auto [d_A, d_U, d_V] = generateRandomLowRankMatrix(handle, gen, m, n, r);
        
        // Compute QR decomposition
        auto [d_Q, d_R, compact] = computeQR(solver_handle, d_A, m, n);
        
        // Generate random matrix X
        hiprandGenerateNormal(gen, d_X, n * batch_size, 0.0f, 1.0f);

        // Time the matrix multiplication
        auto kernel_time = benchmark_kernel([&]() {
            launch_QRX(d_Q, d_R, d_X, d_output_matrix, m, n, r, batch_size);
        });
        total_time_matrix += kernel_time;

        hipFree(d_A);
        hipFree(d_U);
        hipFree(d_V);
    }

    // Print results
    std::cout << "Average time for QRx: " << (total_time_vector / num_trials) * 1000 << " ms\n";
    std::cout << "Average time for QRX: " << (total_time_matrix / num_trials) * 1000 << " ms\n";

    // Cleanup
    hipFree(d_Q);
    hipFree(d_R);
    hipFree(d_x);
    hipFree(d_X);
    hipFree(d_output);
    hipFree(d_output_matrix);
    hipFree(d_temp);
    hipblasDestroy(handle);
    hiprandDestroyGenerator(gen);
    hipsolverDnDestroy(solver_handle);

    return 0;
}
