#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <hipsolver.h>
#include "reference_kernels.cuh"

// Helper function to generate random low-rank matrix
void generateRandomLowRankMatrix(hipblasHandle_t handle, hiprandGenerator_t gen,
                               float* d_A, int m, int n, int rank) {
    // Generate A = UV^T where U is m x rank and V is n x rank
    float* d_U;
    float* d_V;
    hipMalloc(&d_U, m * rank * sizeof(float));
    hipMalloc(&d_V, n * rank * sizeof(float));

    // Generate random U and V
    hiprandGenerateNormal(gen, d_U, m * rank, 0.0f, 1.0f);
    hiprandGenerateNormal(gen, d_V, n * rank, 0.0f, 1.0f);

    // Compute A = U * V^T
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                m, n, rank,
                &alpha,
                d_U, m,
                d_V, n,
                &beta,
                d_A, m);

    hipFree(d_U);
    hipFree(d_V);
}

void computeQR(hipsolverHandle_t solver_handle, float* d_A, float* d_Q, float* d_R, 
               int m, int n) {
    // Allocate workspace for QR decomposition
    int work_size = 0;
    hipsolverDnSgeqrf_bufferSize(solver_handle, m, n, d_A, m, &work_size);
    
    float* d_work;
    float* d_tau;
    int* d_info;
    hipMalloc(&d_work, work_size * sizeof(float));
    hipMalloc(&d_tau, n * sizeof(float));
    hipMalloc(&d_info, sizeof(int));

    // Compute QR factorization (A = QR)
    hipsolverDnSgeqrf(solver_handle, m, n, d_A, m, d_tau, d_work, work_size, d_info);

    // Copy upper triangular part to R
    hipMemset(d_R, 0, n * n * sizeof(float));
    for(int i = 0; i < n; i++) {
        hipMemcpy(&d_R[i * n], &d_A[i * m], (i + 1) * sizeof(float), 
                  hipMemcpyDeviceToDevice);
    }

    // Compute Q explicitly
    hipMemcpy(d_Q, d_A, m * n * sizeof(float), hipMemcpyDeviceToDevice);
    hipsolverDnSorgqr(solver_handle, m, n, n, d_Q, m, d_tau, d_work, work_size, d_info);

    // Cleanup
    hipFree(d_work);
    hipFree(d_tau);
    hipFree(d_info);
}

int main(int argc, char **argv) {
    const int num_trials = 1000;
    const int m = 1024;  // matrix height
    const int n = 512;   // matrix width
    const int r = 32;    // rank
    const int batch_size = 100;  // for matrix X

    // Initialize CUDA resources
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    // Allocate device memory
    float *d_A, *d_Q, *d_R, *d_x, *d_X, *d_output, *d_output_matrix;
    hipMalloc(&d_A, m * n * sizeof(float));
    hipMalloc(&d_Q, m * n * sizeof(float));
    hipMalloc(&d_R, n * n * sizeof(float));
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_X, n * batch_size * sizeof(float));
    hipMalloc(&d_output, m * sizeof(float));
    hipMalloc(&d_output_matrix, m * batch_size * sizeof(float));

    // Benchmark vector multiplication
    double total_time_vector = 0.0;
    for (int trial = 0; trial < num_trials; trial++) {
        // Generate random low-rank matrix
        generateRandomLowRankMatrix(handle, gen, d_A, m, n, r);
        
        // Compute QR decomposition
        computeQR(solver_handle, d_A, d_Q, d_R, m, n);
        
        // Generate random vector x
        hiprandGenerateNormal(gen, d_x, n, 0.0f, 1.0f);

        // Time the vector multiplication
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        
        // Launch your kernel (you'll need to set appropriate grid/block dimensions)
        launch_QRx<<<(m + 255) / 256, 256>>>(d_Q, d_R, d_x, d_output, m, n, r);
        
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        total_time_vector += std::chrono::duration<double>(end - start).count();
    }

    // Benchmark matrix multiplication
    double total_time_matrix = 0.0;
    for (int trial = 0; trial < num_trials; trial++) {
        // Generate random low-rank matrix
        generateRandomLowRankMatrix(handle, gen, d_A, m, n, r);
        
        // Compute QR decomposition
        computeQR(solver_handle, d_A, d_Q, d_R, m, n);
        
        // Generate random matrix X
        hiprandGenerateNormal(gen, d_X, n * batch_size, 0.0f, 1.0f);

        // Time the matrix multiplication
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        
        launch_QRX<<<dim3((m + 15) / 16, (batch_size + 15) / 16), dim3(16, 16)>>>(
            d_Q, d_R, d_X, d_output_matrix, m, n, r, batch_size);
        
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        total_time_matrix += std::chrono::duration<double>(end - start).count();
    }

    // Print results
    std::cout << "Average time for QRx: " << (total_time_vector / num_trials) * 1000 << " ms\n";
    std::cout << "Average time for QRX: " << (total_time_matrix / num_trials) * 1000 << " ms\n";

    // Cleanup
    hipFree(d_A);
    hipFree(d_Q);
    hipFree(d_R);
    hipFree(d_x);
    hipFree(d_X);
    hipFree(d_output);
    hipFree(d_output_matrix);
    hipblasDestroy(handle);
    hiprandDestroyGenerator(gen);
    hipsolverDnDestroy(solver_handle);

    return 0;
}
