#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <functional>
#include <hipsolver.h>

#include "reference_kernels.cuh"
#include "cuda_utils.cuh"

template<typename F>
double benchmark_kernel(F func) {
    CHECK_CUDA(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    func();
    CHECK_CUDA(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
}

void convert_matrix_major(const float* input_d,
                         float* output_d,
                         int m,      // rows
                         int n,      // cols
                         bool to_column_major = true) {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    if (to_column_major) {
        // Converting row-major to column-major
        CHECK_CUBLAS(hipblasSgeam(handle,
                                HIPBLAS_OP_T,    // transpose
                                HIPBLAS_OP_N,    // no-op
                                m, n,           // output dimensions
                                &alpha,
                                input_d,        // input (viewed as n x m by cuBLAS)
                                n,              // leading dimension of input
                                &beta,
                                nullptr,        // no B matrix
                                m,              // ldb (unused)
                                output_d,       // output
                                m));            // leading dimension of output
    } else {
        // Converting column-major to row-major
        CHECK_CUBLAS(hipblasSgeam(handle,
                                HIPBLAS_OP_T,    // transpose
                                HIPBLAS_OP_N,    // no-op
                                n, m,           // output dimensions (swapped)
                                &alpha,
                                input_d,        // input
                                m,              // leading dimension of input
                                &beta,
                                nullptr,        // no B matrix
                                n,              // ldb (unused)
                                output_d,       // output
                                n));            // leading dimension of output
    }

    CHECK_CUBLAS(hipblasDestroy(handle));
}

struct QtKernel {
    std::string name;
    std::function<void(int, int, const float*, float*)> kernel;
    
    QtKernel(const std::string& n,
             std::function<void(int, int, const float*, float*)> k)
        : name(n), kernel(k) {}
};

int main(int argc, char **argv) {
    bool verbose = false;
    bool memory_usage = false;
    for (int i = 1; i < argc; i++) {
        if (std::string(argv[i]) == "--verbose") {
            verbose = true;
            break;
        } else if (std::string(argv[i]) == "--memory-usage") {
            memory_usage = true;
            break;
        }
    }

    const int num_trials = 100;
    // TODO: Fix for larger matrix sizes (e.g., 96)
    const int size_in = 1024;  // matrix size
    constexpr int tilesize = 32;  // tile size
    constexpr int numthreads = 4;  // compile-time constant
    
    // Initialize CUDA resources
    hipsolverHandle_t solver_handle;
    CHECK_CUSOLVER(hipsolverDnCreate(&solver_handle));

    // Allocate device memory
    float *d_matrix_input, *d_matrix, *d_matrix_out, *d_matrix_out_ref, *d_tau;
    CHECK_CUDA(hipMalloc(&d_matrix_input, size_in * size_in * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_matrix, size_in * size_in * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_matrix_out, size_in * size_in * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_matrix_out_ref, size_in * size_in * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_tau, (size_in / tilesize) * size_in * sizeof(float)));

    // Define kernel implementations
    std::vector<QtKernel> kernels = {
        QtKernel("Original (Evelyne)",
            launch_base_applyQt_singletile_evelyne
        ),
        QtKernel("Improved (Lucas)",
            launch_base_applyQt_singletile
        ),
        QtKernel("Reference Implementation",
            // TODO: This is unfair to cuBLAS; should use efficient kernel & in col major
            // TODO: Maybe we could introduce optional preamble and postamble functions, with
            //       a shared pointer between them?
            [&](int size_in, int diag_iter, const float* tau, float* matrix_out) {
                // Allocate memory for column major result
                float* matrix_out_col_major;
                CHECK_CUDA(hipMalloc(&matrix_out_col_major, size_in * size_in * sizeof(float)));
                convert_matrix_major(matrix_out, matrix_out_col_major, size_in, size_in);
                reference_applyQt(size_in, diag_iter, tau, matrix_out_col_major);
                convert_matrix_major(matrix_out_col_major, matrix_out, size_in, size_in, false);
                CHECK_CUDA(hipFree(matrix_out_col_major));
            }
        )
    };

    // Results structure
    struct KernelResults {
        float total_time = 0.0f;
        float max_error = 0.0f;
    };
    std::vector<KernelResults> results(kernels.size());

    // Benchmark loop:
    // - Initialize matrix with random data
    // - Run QR individually for each diagonal block to populate the diagonal tiles and the tau vector
    // - For each row:
    //   - Reference: Apply the Q' matrix to the tiles to the right of the diagonal and store the results
    //   - Custom: Call the kernel for this diagonal iter
    //   - Compare the results
    for (int trial = 0; trial < num_trials; trial++) {
        // Print memory usage at start of each trial
        if (memory_usage) {
            size_t free_byte, total_byte;
            CHECK_CUDA(hipMemGetInfo(&free_byte, &total_byte));
            float free_gb = free_byte / (1024.0 * 1024.0 * 1024.0);
            float total_gb = total_byte / (1024.0 * 1024.0 * 1024.0);
            float used_gb = total_gb - free_gb;
            printf("\nTrial %d - Memory Usage: Used = %.2f GB, Free = %.2f GB, Total = %.2f GB\n",
               trial, used_gb, free_gb, total_gb);
        }

        // Initialize matrix with random data
        hiprandGenerator_t gen;
        CHECK_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL + trial));
        CHECK_CURAND(hiprandGenerateNormal(gen, d_matrix_input, size_in * size_in, 0.0f, 1.0f));
        CHECK_CURAND(hiprandDestroyGenerator(gen));

        // Run QR individually for each diagonal block to populate the diagonal tiles and the tau vector
        for (int diag_iter = 0; diag_iter < size_in/tilesize - 1; diag_iter++) {
            // Extract the diagonal tile
            float* diag_tile;
            CHECK_CUDA(hipMalloc(&diag_tile, tilesize * tilesize * sizeof(float)));
            
            // Copy the diagonal tile from the matrix
            for(int j = 0; j < tilesize; j++) {
                CHECK_CUDA(hipMemcpy(diag_tile + j * tilesize,
                                    d_matrix_input + (diag_iter * tilesize + j) * size_in + diag_iter * tilesize,
                                    tilesize * sizeof(float),
                                    hipMemcpyDeviceToDevice));
            }
            
            // Perform QR on this tile
            int* devInfo;
            CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
            
            // Query workspace size
            int lwork;
            CHECK_CUSOLVER(hipsolverDnSgeqrf_bufferSize(
                solver_handle,
                tilesize, tilesize,
                diag_tile,
                tilesize,
                &lwork));
                
            // Allocate workspace
            float* workspace;
            CHECK_CUDA(hipMalloc(&workspace, lwork * sizeof(float)));
            
            // Compute QR factorization
            CHECK_CUSOLVER(hipsolverDnSgeqrf(
                solver_handle,
                tilesize, tilesize,
                diag_tile,
                tilesize,
                d_tau + diag_iter * size_in,  // TODO: Clarify why this is the right format for tau
                workspace,
                lwork,
                devInfo));
                
            // Copy the result back to the matrix
            for(int j = 0; j < tilesize; j++) {
                CHECK_CUDA(hipMemcpy(d_matrix_input + (diag_iter * tilesize + j) * size_in + diag_iter * tilesize,
                                    diag_tile + j * tilesize,
                                    tilesize * sizeof(float),
                                    hipMemcpyDeviceToDevice));
            }
            
            // Cleanup
            CHECK_CUDA(hipFree(workspace));
            CHECK_CUDA(hipFree(devInfo));
            CHECK_CUDA(hipFree(diag_tile));
        }

        // For each diagonal block
        for (int diag_iter = 0; diag_iter < size_in/tilesize - 1; diag_iter++) {
            // Copy fresh matrix for each implementation, i.e., discard previous changes
            CHECK_CUDA(hipMemcpy(d_matrix, d_matrix_input, 
                                size_in * size_in * sizeof(float), 
                                hipMemcpyDeviceToDevice));

            // Compute reference results for this diagonal iter
            CHECK_CUDA(hipMemcpy(d_matrix_out_ref, d_matrix,
                                size_in * size_in * sizeof(float),
                                hipMemcpyDeviceToDevice));
            reference_applyQt(size_in, diag_iter, d_tau, d_matrix_out_ref);

            std::vector<float> host_ref(size_in * size_in);
            CHECK_CUDA(hipMemcpy(host_ref.data(), d_matrix_out_ref,
                                size_in * size_in * sizeof(float),
                                hipMemcpyDeviceToHost));
            
            // Test each implementation
            for (size_t i = 0; i < kernels.size(); i++) {
                // Copy input from column major (default cuBLAS format) to row major
                convert_matrix_major(d_matrix, d_matrix_out, size_in, size_in);

                auto& kernel = kernels[i];
                auto& result = results[i];
                
                // Time the kernel
                result.total_time += benchmark_kernel([&]() {
                    kernel.kernel(size_in, diag_iter, d_tau, d_matrix_out);
                });
                
                // Copy results to host for comparison
                std::vector<float> host_custom(size_in * size_in);
                CHECK_CUDA(hipMemcpy(host_custom.data(), d_matrix_out,
                                    size_in * size_in * sizeof(float),
                                    hipMemcpyDeviceToHost));
                
                // Compare only the relevant tiles. We include the diagonal tiles since these should
                // not be modified by the kernel
                for (int tile = diag_iter; tile < size_in/tilesize; tile++) {
                    for (int row = diag_iter * tilesize; row < (diag_iter + 1) * tilesize; row++) {
                        for (int col = tile * tilesize; col < (tile + 1) * tilesize; col++) {
                            int col_major_idx = col * size_in + row;
                            int row_major_idx = row * size_in + col;
                            float diff = std::abs(host_custom[row_major_idx] - host_ref[col_major_idx]);
                            result.max_error = std::max(result.max_error, diff);

                            // std::cout << "tile: " << tile << " row: " << row << " col: " << col << "\n";
                            // std::cout << "host_custom[" << row_major_idx << "] = " << host_custom[row_major_idx] << "\n";
                            // std::cout << "host_ref[" << col_major_idx << "] = " << host_ref[col_major_idx] << "\n";

                            if (diff > 1e-5 && verbose) {
                                std::cout << "Large difference at tile (" 
                                        << row/tilesize << "," << col/tilesize 
                                        << ") rel_position (" << row % tilesize << "," << col % tilesize 
                                        << ") abs_position (" << row << "," << col << ")\n"
                                        << "): custom=" << host_custom[row_major_idx] 
                                        << " ref=" << host_ref[col_major_idx] 
                                        << " diff=" << diff << "\n";
                            }
                        }
                    }
                }
            }
        }
    }
    
    // Print results
    std::cout << "\nResults (averaged over " << num_trials << " trials) for (" << size_in << "x" << size_in << "):\n";
    std::cout << std::string(60, '-') << "\n";
    std::cout << std::setw(30) << "Implementation" 
              << std::setw(15) << "Time (ms)" 
              << std::setw(15) << "Max Error" << "\n";
    std::cout << std::string(60, '-') << "\n";
    
    for (size_t i = 0; i < kernels.size(); i++) {
        std::cout << std::setw(30) << kernels[i].name 
                  << std::setw(15) << results[i].total_time / num_trials / 1000.0f
                  << std::setw(15) << results[i].max_error << "\n";
    }
    
    // Cleanup
    CHECK_CUDA(hipFree(d_matrix_input));
    CHECK_CUDA(hipFree(d_matrix));
    CHECK_CUDA(hipFree(d_matrix_out));
    CHECK_CUDA(hipFree(d_matrix_out_ref));
    CHECK_CUDA(hipFree(d_tau));
    CHECK_CUSOLVER(hipsolverDnDestroy(solver_handle));

    return 0;
}